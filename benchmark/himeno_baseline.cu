

#define NN 750
#define BLOCKSIZEX
#define BLOCKSIZEY 
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX
#define GRIDSIZEY
#define GRIDSIZE GRIDSIZEX * GRIDSIZEY
#define MIMAX 128
#define MJMAX BLOCKSIZEY * GRIDSIZEY 
#define MKMAX BLOCKSIZEX * GRIDSIZEX
#define THREAD_NUM BLOCKSIZE * GRIDSIZE

/*static float p[MIMAX][MJMAX][MKMAX];
static float a[MIMAX][MJMAX][MKMAX][4];
static float b[MIMAX][MJMAX][MKMAX][3];
static float c[MIMAX][MJMAX][MKMAX][3];
static float bnd[MIMAX][MJMAX][MKMAX];
static float work1[MIMAX][MJMAX][MKMAX];
static float work2[MIMAX][MJMAX][MKMAX];*/

static int imax, jmax, kmax;
static float omega;

void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, n;
	float s0, ss, temp;
	const int tid = threadIdx.x;
	const int size = (imax-1)/(imax-1);

	for(n=0;n<nn;++n){
		
	temp=0.0;

		for(i=tid*size ; i<(tid+1)*size ; ++i){
					s0 = a0[i*jmax*kmax+j*kmax+k] * p[(i+1)*jmax*kmax+j*kmax+k]
					   + a1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j+1)*kmax+k]
					   + a2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k+1)]
					   + b0[i*jmax*kmax+j*kmax+k] * ( p[(i+1)*jmax*kmax+(j+1)*kmax+k] - p[(i+1)*jmax*kmax+(j-1)*kmax+k]
						       - p[(i-1)*jmax*kmax+(j+1)*kmax+k] + p[(i-1)*jmax*kmax+(j-1)*kmax+k] )
					   + b1[i*jmax*kmax+j*kmax+k] * ( p[i*jmax*kmax+(j+1)*kmax+(k+1)] - p[i*jmax*kmax+(j-1)*kmax+(k+1)]
						       - p[i*jmax*kmax+(j+1)*kmax+(k-1)] + p[i*jmax*kmax+(j-1)*kmax+(k-1)] )
					   + b2[i*jmax*kmax+j*kmax+k] * ( p[(i+1)*jmax*kmax+j*kmax+(k+1)] - p[(i-1)*jmax*kmax+j*kmax+(k+1)]
						       - p[(i+1)*jmax*kmax+j*kmax+(k-1)] + p[(i-1)*jmax*kmax+j*kmax+(k-1)] )
					   + c0[i*jmax*kmax+j*kmax+k] * p[(i-1)*jmax*kmax+j*kmax+k]
					   + c1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j-1)*kmax+k]
					   + c2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k-1)]
					   + wrk1[i*jmax*kmax+j*kmax+k];

					ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

					temp = temp + ss*ss;

					wrk2[i*jmax*kmax+j*kmax+k] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
		}

	for(i=tid*size ; i<(tid+1)*size ; ++i)
		for(j=1 ; j<jmax-1 ; ++j)
			for(k=1 ; k<kmax-1 ; ++k)
				p[i*jmax*kmax+j*kmax+k] = wrk2[i*jmax*kmax+j*kmax+k];
	} /* end n loop */

	gosa[tid] = temp;
}


int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[THREAD_NUM];

	/************************************/
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	/************************************/
	imax = MIMAX-1;
	jmax = MJMAX-1;
	kmax = MKMAX-1;
	//int N_IJK = MIMAX*MJMAX*MKMAX;
	int N_IJK = imax*jmax*kmax;
	/************************************/
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	/************************************/


	omega = 0.8;

	//initial_maxtrix();

	/******allocate mem on CPU***********/
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*N_IJK);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	/************************************/

	/******allocate mem on GPU***********/
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));
	
	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*THREAD_NUM);
	/************************************/

	/*****Initialize*********************/
	//int i,j,k;

	for(i=0 ; i<imax ; ++i)
		for(j=0 ; j<jmax ; ++j)
			for(k=0 ; k<kmax ; ++k){
				a0[i*jmax*kmax+j*kmax+k]=0.0;
				a1[i*jmax*kmax+j*kmax+k]=0.0;
				a2[i*jmax*kmax+j*kmax+k]=0.0;
				a3[i*jmax*kmax+j*kmax+k]=0.0;
				b0[i*jmax*kmax+j*kmax+k]=0.0;
				b1[i*jmax*kmax+j*kmax+k]=0.0;
				b2[i*jmax*kmax+j*kmax+k]=0.0;
				c0[i*jmax*kmax+j*kmax+k]=0.0;
				c1[i*jmax*kmax+j*kmax+k]=0.0;
				c2[i*jmax*kmax+j*kmax+k]=0.0;
				p[i*jmax*kmax+j*kmax+k]=0.0;
				wrk1[i*jmax*kmax+j*kmax+k]=0.0;
				bnd[i*jmax*kmax+j*kmax+k]=0.0;
	      		}

	for(i=0 ; i<imax ; ++i)
		for(j=0 ; j<jmax ; ++j)
			for(k=0 ; k<kmax ; ++k){
				a0[i*jmax*kmax+j*kmax+k]=1.0;
				a1[i*jmax*kmax+j*kmax+k]=1.0;
				a2[i*jmax*kmax+j*kmax+k]=1.0;
				a3[i*jmax*kmax+j*kmax+k]=1.0/6.0;
				b0[i*jmax*kmax+j*kmax+k]=0.0;
				b1[i*jmax*kmax+j*kmax+k]=0.0;
				b2[i*jmax*kmax+j*kmax+k]=0.0;
				c0[i*jmax*kmax+j*kmax+k]=1.0;
				c1[i*jmax*kmax+j*kmax+k]=1.0;
				c2[i*jmax*kmax+j*kmax+k]=1.0;
				p[i*jmax*kmax+j*kmax+k]=(float)(k*k)/(float)((kmax-1)*(kmax-1));
				wrk1[i*jmax*kmax+j*kmax+k]=0.0;
				bnd[i*jmax*kmax+j*kmax+k]=1.0;
			}
	/************************************/

	/*****copy array to device mem*******/
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//hipMemcpy(dev_gosa, gosa, sizeof(float), hipMemcpyHostToDevice);
	/************************************/

	cpu0 = second(); /**measuring**/

	jacobi(1,THREAD_NUM, dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, NN, imax, jmax, kmax, omega, dev_gosa);

	//hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*THREAD_NUM, hipMemcpyDeviceToHost);

	/******Free mem on the GPU**********/
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	/************************************/

	/********Final sum of gosa***********/
	for(int gosa_index=0; gosa_index<THREAD_NUM; gosa_index++){
		final_gosa += gosa[gosa_index];
	}
	/************************************/

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	//if(cpu1 != 0.0)
	//	xmflops2 = nflop/cpu1*1.0e-6*(float)NN;

	//score = xmflops2/32.27;

	return(0);

}