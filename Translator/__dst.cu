#include "hip/hip_runtime.h"
# 1 "__cp.cu"
# 1 "<built-in>" 1
# 1 "<built-in>" 3
# 330 "<built-in>" 3
# 1 "<command line>" 1
# 1 "<built-in>" 2
# 1 "__cp.cu" 2
# 15 "__cp.cu"
























static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;

double second(){
	struct timeval tm;
	double t;

	static int base_sec = 0, base_usec = 0;

	gettimeofday(&tm, NULL);

	if(base_sec == 0 && base_usec == 0){
		base_sec = tm.tv_sec;
		base_usec = tm.tv_usec;
		t = 0.0;
	}
	else{
		t = (double)(tm.tv_sec-base_sec) + ((double)(tm.tv_usec-base_usec))/1.0e6;
	}

	return t;
}


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, j2, k2, n, xy, c, csb;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
	k = threadIdx.x + (blockDim.x-2) * blockIdx.x + 1;
	j = threadIdx.y + (blockDim.y-2) * blockIdx.y + 1;
	k2 = threadIdx.x + blockDim.x * blockIdx.x;
	j2 = threadIdx.y + blockDim.y * blockIdx.y;
	const int tid = (k-1) + (j-1) * (kmax-2);
	xy = kmax * jmax;
	__shared__ float sb[6 * 4];
	csb = threadIdx.x + threadIdx.y * blockDim.x;
	for(n=0;n<nn;++n){
		c = j * kmax + k;
		temp=0.0;
		for(i=1 ; i<imax-1 ; ++i){
				syncthreads();
			if(0 < threadIdx.x && k < kmax-1 && 0 < j && j < jmax-1){
				s0 = a0[i*jmax*kmax+j*kmax+k] * p[(i+1)*jmax*kmax+j*kmax+k]
				+ a1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j+1)*kmax+k]
				+ a2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k+1)]
				+ b0[i*jmax*kmax+j*kmax+k] 
					*(p[(i+1)*jmax*kmax+(j+1)*kmax+k] 
					- p[(i+1)*jmax*kmax+(j-1)*kmax+k]
					- p[(i-1)*jmax*kmax+(j+1)*kmax+k] 
					+ p[(i-1)*jmax*kmax+(j-1)*kmax+k] )
				+ b1[i*jmax*kmax+j*kmax+k] 
					*(p[i*jmax*kmax+(j+1)*kmax+(k+1)] 
					- p[i*jmax*kmax+(j-1)*kmax+(k+1)]
					- p[i*jmax*kmax+(j-1)*kmax+(k-1)]
					+ p[i*jmax*kmax+(j+1)*kmax+(k-1)])
				+ b2[i*jmax*kmax+j*kmax+k] 
					*(p[(i+1)*jmax*kmax+j*kmax+(k+1)] 
					- p[(i-1)*jmax*kmax+j*kmax+(k+1)]
					- p[(i+1)*jmax*kmax+j*kmax+(k-1)] 
					+ p[(i-1)*jmax*kmax+j*kmax+(k-1)] )
				+ c0[i*jmax*kmax+j*kmax+k] * p[(i-1)*jmax*kmax+j*kmax+k]
				+ c1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j-1)*kmax+k]
				+ c2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k-1)]
				+ wrk1[i*jmax*kmax+j*kmax+k];

				ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

				temp = temp + ss*ss;

				wrk2[i*6 * 4 * 3 * 6 +j2*6*3 +k2] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
				c += xy;
			}
		}
		syncthreads();
		if(0 < threadIdx.x && threadIdx.x < blockDim.x-1 && 0 < threadIdx.y && threadIdx.y < blockDim.y-1){
			for(i=1; i<imax-1; i++){
				s0 = a0[i*jmax*kmax+j*kmax+k] * wrk2[(i+1)*6 * 4 * 3 * 6 +j2*6*3 +k2]
				+ a1[i*jmax*kmax+j*kmax+k] * wrk2[i*6 * 4 * 3 * 6 +(j2+1)*6*3 +k2]
				+ a2[i*jmax*kmax+j*kmax+k] * wrk2[i*6 * 4 * 3 * 6 +j2*6*3 +(k2+1)]
				+ b0[i*jmax*kmax+j*kmax+k] 
					*(wrk2[(i+1)*6 * 4 * 3 * 6 +(j2+1)*6*3 +k2] 
					- wrk2[(i+1)*6 * 4 * 3 * 6 +(j2-1)*6*3 +k2]
					- wrk2[(i-1)*6 * 4 * 3 * 6 +(j2+1)*6*3 +k2] 
					+ wrk2[(i-1)*6 * 4 * 3 * 6 +(j2-1)*6*3 +k2] )
				+ b1[i*jmax*kmax+j*kmax+k] 
					*(wrk2[i*6 * 4 * 3 * 6 +(j2+1)*6*3 +(k2+1)] 
					- wrk2[i*6 * 4 * 3 * 6 +(j2-1)*6*3 +(k2+1)]
					- wrk2[i*6 * 4 * 3 * 6 +(j2-1)*6*3 +(k2-1)]
					+ wrk2[i*6 * 4 * 3 * 6 +(j2+1)*6*3 +(k2-1)])
				+ b2[i*jmax*kmax+j*kmax+k] 
					*(wrk2[(i+1)*6 * 4 * 3 * 6 +j2*6*3 +(k2+1)] 
					- wrk2[(i-1)*6 * 4 * 3 * 6 +j2*6*3 +(k2+1)]
					- wrk2[(i+1)*6 * 4 * 3 * 6 +j2*6*3 +(k2-1)] 
					+ wrk2[(i-1)*6 * 4 * 3 * 6 +j2*6*3 +(k2-1)] )
				+ c0[i*jmax*kmax+j*kmax+k] * wrk2[(i-1)*6 * 4 * 3 * 6 +j2*6*3 +k2]
				+ c1[i*jmax*kmax+j*kmax+k] * wrk2[i*6 * 4 * 3 * 6 +(j2-1)*6*3 +k2]
				+ c2[i*jmax*kmax+j*kmax+k] * wrk2[i*6 * 4 * 3 * 6 +j2*6*3 +(k2-1)]
				+ wrk1[i*jmax*kmax+j*kmax+k];

				ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - wrk2[i*6 * 4 * 3 * 6 +j2*6*3 +k2] ) * bnd[i*jmax*kmax+j*kmax+k];

				temp = temp + ss*ss;

				p[i*jmax*kmax+j*kmax+k] = wrk2[i*6 * 4 * 3 * 6 +j2*6*3 +k2] + omega * ss;
				c += xy;
			}
		}	
	} 
	syncthreads();
	gosa[tid] = temp;
}

int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[6 * 4 * 3 * 6];

	
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	
	mimax = 4;
	mjmax = 6 * (4 - 2) + 2;
	mkmax = 3 * (6 - 2) + 2;
	imax = 4 -1;
	jmax = 6 * (4 - 2) + 2 -1;
	kmax = 3 * (6 - 2) + 2 -1;
	//int N_IJK = 4*6 * (4 - 2) + 2*3 * (6 - 2) + 2;
	int N_IJK = mimax*mjmax*mkmax;
	int WORKSIZE = 6 * 4 * 3 * 6*mimax;
	
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	


	omega = 0.8;

	//initial_maxtrix();

	
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*WORKSIZE);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	

	
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, WORKSIZE*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*6 * 4 * 3 * 6);
	

	
	//int i,j,k;
	
# 259 "__cp.cu"

	for(i=0 ; i<mimax ; ++i){
		for(j=0 ; j<mjmax ; ++j){
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				p[i*mjmax*mkmax+j*mkmax+k]=(float)(i*i)/(float)(imax*imax);
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=1.0;
			}
		}
	}
	

	
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, WORKSIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//hipMemcpy(dev_gosa, gosa, sizeof(float), hipMemcpyHostToDevice);
	

	printf("mimax = %d mjmax = %d mkmax = %d\n", 4, 6 * (4 - 2) + 2, 3 * (6 - 2) + 2);
	printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	cpu0 = second(); 

	dim3 block(6, 4, 1);
	dim3 grid(3, 6, 1);

	jacobi<<<grid, block>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, 3, mimax, mjmax, mkmax, omega, dev_gosa);

	hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*6 * 4 * 3 * 6, hipMemcpyDeviceToHost);

	
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	

	
	for(int gosa_index=0; gosa_index<6 * 4 * 3 * 6; gosa_index++){
		//printf("%f\n", gosa[gosa_index]);
		final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
	}
	

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	if(cpu1 != 0.0){
		xmflops2 = nflop/cpu1*1.0e-6*(float)3;
	}

	score = xmflops2/32.27;

	printf("gpu: %f sec.\n", cpu1);
	printf("Loop executed for %d times\n", 3);
	printf("Gosa: %e \n", final_gosa);
	//printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

	return(0);
}

