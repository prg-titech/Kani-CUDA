#include "hip/hip_runtime.h"
# 1 "__cp.cu"
# 1 "<built-in>" 1
# 1 "<built-in>" 3
# 360 "<built-in>" 3
# 1 "<command line>" 1
# 1 "<built-in>" 2
# 1 "__cp.cu" 2























static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;



# 65 "__cp.cu"



# 87 "__cp.cu"


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	profile("threadIdx.x threadIdx.y blockDim.x blockDim.y csb SIZE i j");
	int i, j, k, n, xy, c, csb;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
	k = threadIdx.x + blockDim.x * blockIdx.x + 1;
	j = threadIdx.y + blockDim.y * blockIdx.y + 1;
	const int tid = (k-1) + (j-1) * (kmax-2);
	xy = kmax * jmax;
	const int SIZE = blockDim.x * blockDim.y;
	__shared__ float sb[SIZE];
	csb = threadIdx.x + threadIdx.y * blockDim.x;
	for(n=0;n<nn;++n){
		c = xy + j * kmax + k;
		temp=0.0;
		for(i=1 ; i<imax-1 ; ++i){
			sb[csb] = p[c];
			syncthreads();
      //printf("shared: %f\n", sb[csb]);
      //syncthreads();
			s0 = a0[i*jmax*kmax+j*kmax+k] * p[(i+1)*jmax*kmax+j*kmax+k]
			+ a1[i*jmax*kmax+j*kmax+k] * __opt__.p[i*jmax*kmax+(j+1)*kmax+k]
			+ a2[i*jmax*kmax+j*kmax+k] * __opt__.p[i*jmax*kmax+j*kmax+(k+1)]
			+ b0[i*jmax*kmax+j*kmax+k] * ( 
				p[(i+1)*jmax*kmax+(j+1)*kmax+k] 
				- p[(i+1)*jmax*kmax+(j-1)*kmax+k]
				- p[(i-1)*jmax*kmax+(j+1)*kmax+k] 
				+ p[(i-1)*jmax*kmax+(j-1)*kmax+k] )
			+ b1[i*jmax*kmax+j*kmax+k] *(
				__opt__.p[i*jmax*kmax+(j+1)*kmax+(k+1)]
				- __opt__.p[i*jmax*kmax+(j-1)*kmax+(k+1)]
				- __opt__.p[i*jmax*kmax+(j-1)*kmax+(k-1)]
				+ __opt__.p[i*jmax*kmax+(j+1)*kmax+(k-1)])
			+ b2[i*jmax*kmax+j*kmax+k] * ( p[(i+1)*jmax*kmax+j*kmax+(k+1)] - p[(i-1)*jmax*kmax+j*kmax+(k+1)]
				- p[(i+1)*jmax*kmax+j*kmax+(k-1)] + p[(i-1)*jmax*kmax+j*kmax+(k-1)] )
			+ c0[i*jmax*kmax+j*kmax+k] * p[(i-1)*jmax*kmax+j*kmax+k]
			+ c1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j-1)*kmax+k]
			+ c2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k-1)]
			+ wrk1[i*jmax*kmax+j*kmax+k];

			ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

			temp = temp + ss*ss;

			wrk2[i*jmax*kmax+j*kmax+k] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
			c += xy;
		}
		syncthreads();
		for(i=1 ; i<imax-1 ; ++i){
			p[i*jmax*kmax+j*kmax+k] = wrk2[i*jmax*kmax+j*kmax+k];
		}
  	} 
		syncthreads();
  //printf("%d: p[%d] = %d\n", tid,i*jmax*kmax+j*kmax+k,p[i*jmax*kmax+j*kmax+k]);
  //printf("shared: %f", sb[csb]);
		gosa[tid] = temp;
	}

	int main(){
		int i, j, k;
		float final_gosa;
		double cpu0, cpu1, nflop, xmflops2, score;

		float gosa[4 * 4 * 2 * 2];

	
		float *p;
		float *a0, *a1, *a2, *a3;
		float *b0, *b1, *b2;
		float *c0, *c1, *c2;

		float *bnd;
		float *wrk1, *wrk2;
	
		mimax = 3;
		mjmax = 2 * 4 + 2;
		mkmax = 2 * 4 + 2;
		imax = 3 -1;
		jmax = 2 * 4 + 2 -1;
		kmax = 2 * 4 + 2 -1;
	//int N_IJK = 3*2 * 4 + 2*2 * 4 + 2;
		int N_IJK = mimax*mjmax*mkmax;
	
		float *dev_p;
		float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
		float *dev_b0, *dev_b1, *dev_b2;
		float *dev_c0, *dev_c1, *dev_c2;

		float *dev_bnd;
		float *dev_wrk1, *dev_wrk2;

		float *dev_gosa;
	


		omega = 0.8;

	//initial_maxtrix();

	
		a0 = (float*)malloc(sizeof(float)*N_IJK);
		a1 = (float*)malloc(sizeof(float)*N_IJK);
		a2 = (float*)malloc(sizeof(float)*N_IJK);
		a3 = (float*)malloc(sizeof(float)*N_IJK);

		b0 = (float*)malloc(sizeof(float)*N_IJK);
		b1 = (float*)malloc(sizeof(float)*N_IJK);
		b2 = (float*)malloc(sizeof(float)*N_IJK);

		c0 = (float*)malloc(sizeof(float)*N_IJK);
		c1 = (float*)malloc(sizeof(float)*N_IJK);
		c2 = (float*)malloc(sizeof(float)*N_IJK);

		p = (float*)malloc(sizeof(float)*N_IJK);

		wrk1 = (float*)malloc(sizeof(float)*N_IJK);
		wrk2 = (float*)malloc(sizeof(float)*N_IJK);
		bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	

	
		hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

		hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

		hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

		hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

		hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
		hipMalloc((void**)&dev_wrk2, N_IJK*sizeof(float));

		hipMalloc((void**)&dev_gosa, sizeof(float)*4 * 4 * 2 * 2);
	

	
	//int i,j,k;

		for(i=0 ; i<mimax ; ++i){
			for(j=0 ; j<mjmax ; ++j){
				for(k=0 ; k<mkmax ; ++k){
					a0[i*mjmax*mkmax+j*mkmax+k]=0.0;
					a1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					a2[i*mjmax*mkmax+j*mkmax+k]=0.0;
					a3[i*mjmax*mkmax+j*mkmax+k]=0.0;
					b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
					b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
					c0[i*mjmax*mkmax+j*mkmax+k]=0.0;
					c1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					c2[i*mjmax*mkmax+j*mkmax+k]=0.0;
					p[i*mjmax*mkmax+j*mkmax+k]=0.0;
					wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					bnd[i*mjmax*mkmax+j*mkmax+k]=0.0;
				}
			}
		}

		for(i=0 ; i<mimax ; ++i){
			for(j=0 ; j<mjmax ; ++j){
				for(k=0 ; k<mkmax ; ++k){
					a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
					a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
					a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
					a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
					b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
					b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
					c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
					c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
					c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
					p[i*mjmax*mkmax+j*mkmax+k]=(float)i*mjmax*mkmax+j*mkmax+k;
					wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
					bnd[i*mjmax*mkmax+j*mkmax+k]=1.0;
				}
			}
		}
	

	
		hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_wrk2, wrk2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//hipMemcpy(dev_gosa, gosa, sizeof(float), hipMemcpyHostToDevice);
	

		printf("mimax = %d mjmax = %d mkmax = %d\n", 3, 2 * 4 + 2, 2 * 4 + 2);
		printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	//cpu0 = second(); 

		dim3 block(4, 4, 1);
		dim3 grid(2, 2, 1);

		jacobi<<<grid, block>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, 3, mimax, mjmax, mkmax, omega, dev_gosa);

		hipDeviceSynchronize();

	//cpu1 = second();

		hipMemcpy(&gosa, dev_gosa, sizeof(float)*4 * 4 * 2 * 2, hipMemcpyDeviceToHost);

	
		hipFree(dev_a0);
		hipFree(dev_a1);
		hipFree(dev_a2);
		hipFree(dev_a3);
		hipFree(dev_b0);
		hipFree(dev_b1);
		hipFree(dev_b2);
		hipFree(dev_c0);
		hipFree(dev_c1);
		hipFree(dev_c2);
		hipFree(dev_p);
		hipFree(dev_wrk1);
		hipFree(dev_wrk2);
		hipFree(dev_bnd);

		hipFree(dev_gosa);
	

	
		for(int gosa_index=0; gosa_index<4 * 4 * 2 * 2; gosa_index++){
			final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
		}
	

		nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	//	if(cpu1 != 0.0)
	//		xmflops2 = nflop/cpu1*1.0e-6*(float)3;

		score = xmflops2/32.27;

		printf("gpu: %f sec.\n", cpu1);
		printf("Loop executed for %d times\n", 3);
		printf("Gosa: %e \n", final_gosa);
	//printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

		return(0);

	}

