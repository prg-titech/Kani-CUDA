#include "hip/hip_runtime.h"
# 1 "__cp.cu"
# 1 "<built-in>" 1
# 1 "<built-in>" 3
# 360 "<built-in>" 3
# 1 "<command line>" 1
# 1 "<built-in>" 2
# 1 "__cp.cu" 2
__global__ void stencil_kernel(int *in, int *out, int nx, int ny){
    int i, j, i2, j2, c, c2, csb;
    i = blockIdx.x * (blockDim.x - 2) + threadIdx.x - 1;
    i = max(0, i);
    i = min(i, nx - 1);
    j = blockIdx.y * (blockDim.y - 2) + threadIdx.y - 1;
    j = max(0, j);
    j = max(j, ny - 1);
    c = nx * j + i;
    float sb[threadDim.x * threadDim.y];
    csb = blockDim.x * threadIdx.y + threadIdx.x;
    sb[csb] = in[(i == 0) ? c : c - 1]
        + in[(i == nx - 1) ? c : c + 1] 
        + in[(j == 0) ? c : c - nx] 
        + in[(j == ny - 1) ? c : c + nx] 
        + in[c];
    syncthreads();
    i2 = blockIdx.x * (blockDim.x-2) + min(threadIdx.x, blockDim.x-3);
    i2 = min(i2, nx - 1);
    j2 = blockIdx.y * (blockDim.y-2) + min(threadIdx.y, blockDim.y-3);
    j2 = min(j2, ny -1);
    c2 = nx * j2 + i2;
    csb2 = blockDim.x * (j2%(bockDim.y-2) + 1) + i2%(blockDim.x-2) + 1;
    out[c2] = sb[(i2 == 0) ? scb2 : scb2 - 1]
        + sb[(i2 == nx - 1) ? scb2 : scb2 + 1] 
        + sb[(j2 == 0) ? scb2 : scb2 - nx] 
        + sb[(j2 == ny - 1) ? scb2 : scb2 + nx] 
        + sb[scb2];}

