
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>

#define BLOCKSIZEX 96
#define BLOCKSIZEY 4
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX 4
#define GRIDSIZEY 64
#define GRIDSIZE GRIDSIZEX * GRIDSIZEY
#define THREAD_NUM BLOCKSIZE * GRIDSIZE

#define MIMAX	256
#define MJMAX	GRIDSIZEY * BLOCKSIZEY + 2
#define MKMAX	GRIDSIZEX * BLOCKSIZEX + 2

#define NN 750

/*static float p[MIMAX][MJMAX][MKMAX];
static float a[MIMAX][MJMAX][MKMAX][4];
static float b[MIMAX][MJMAX][MKMAX][3];
static float c[MIMAX][MJMAX][MKMAX][3];
static float bnd[MIMAX][MJMAX][MKMAX];
static float work1[MIMAX][MJMAX][MKMAX];
static float work2[MIMAX][MJMAX][MKMAX];*/

static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;

double second(){
	struct timeval tm;
	double t;

	static int base_sec = 0, base_usec = 0;

	gettimeofday(&tm, NULL);

	if(base_sec == 0 && base_usec == 0){
		base_sec = tm.tv_sec;
		base_usec = tm.tv_usec;
		t = 0.0;
	}
	else{
		t = (double)(tm.tv_sec-base_sec) + ((double)(tm.tv_usec-base_usec))/1.0e6;
	}

	return t;
}


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, n, xy, c, csb;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
	k = threadIdx.x + blockDim.x * blockIdx.x + 1;
	j = threadIdx.y + blockDim.y * blockIdx.y + 1;
	const int tid = (k-1) + (j-1) * (kmax-2);
	xy = kmax * jmax;
	extern __shared__ float sb[];
  float *sb_t = sb;
  float *sb_m = sb + (blockDim.x*blockDim.y);
  float *sb_b = sb + 2 * (blockDim.x*blockDim.y);
	csb = threadIdx.x + threadIdx.y * blockDim.x;
	for(n=0;n<nn;++n){
		c = j * kmax + k;
		temp = 0.0;
		sb_m[csb] = p[c];
		sb_b[csb] = p[c+xy];
		for(i=1 ; i<imax-1 ; ++i){
			c += xy;
      float *sb_tmp = sb_t;
			sb_t = sb_m;
			sb_m = sb_b;
      sb_b = sb_tmp;
			sb_b[csb] = p[c+xy];
			//printf("shared: %f\n", sb_b[csb]);
			__syncthreads();
			s0 =
				a0[i*jmax*kmax+j*kmax+k] * sb_b[csb]
				+ a1[i*jmax*kmax+j*kmax+k] * (!(threadIdx.y==blockDim.y-1) ? sb_m[csb + blockDim.x] : p[i*jmax*kmax+(j+1)*kmax+k])
				+ a2[i*jmax*kmax+j*kmax+k] * (!(threadIdx.x==blockDim.x-1) ? sb_m[csb + 1] : p[i*jmax*kmax+j*kmax+(k+1)])
				+ b0[i*jmax*kmax+j*kmax+k] * (
				  (!(threadIdx.y==blockDim.y-1) ? sb_b[csb + blockDim.x] : p[(i+1)*jmax*kmax+(j+1)*kmax+k])
				- (!(threadIdx.y==0) ? sb_b[csb - blockDim.x] : p[(i+1)*jmax*kmax+(j-1)*kmax+k])
				- (!(threadIdx.y==blockDim.y-1) ? sb_t[csb + blockDim.x] : p[(i-1)*jmax*kmax+(j+1)*kmax+k])
				+ (!(threadIdx.y==0) ? sb_t[csb - blockDim.x] : p[(i-1)*jmax*kmax+(j-1)*kmax+k]) )
				+ b1[i*jmax*kmax+j*kmax+k] *(
				  ((!(threadIdx.x==(blockDim.x - 1))&&!(threadIdx.y==(blockDim.y - 1))) ? sb_m[csb + blockDim.x + 1] : p[i*jmax*kmax+(j+1)*kmax+(k+1)])
				- ((!(threadIdx.y==0)&&!(threadIdx.x==(blockDim.x - 1))) ? sb_m[csb - blockDim.x + 1] : p[i*jmax*kmax+(j-1)*kmax+(k+1)])
				- ((!(threadIdx.y==0)&&!(threadIdx.x==0)) ? sb_m[csb - blockDim.x - 1] : p[i*jmax*kmax+(j-1)*kmax+(k-1)])
				+ ((!(threadIdx.x==0)&&!(threadIdx.y==(blockDim.y - 1))) ? sb_m[csb + blockDim.x - 1] : p[i*jmax*kmax+(j+1)*kmax+(k-1)]))
				+ b2[i*jmax*kmax+j*kmax+k] *(
				  ( !(threadIdx.x==(blockDim.x - 1)) ? sb_b[1 + csb] : p[(i+1)*jmax*kmax+j*kmax+(k+1)] )
				- ( !(threadIdx.x==(blockDim.x - 1)) ? sb_t[csb + 1] : p[(i-1)*jmax*kmax+j*kmax+(k+1)] )
				- ( !(threadIdx.x==0) ? sb_b[csb - 1] : p[(i+1)*jmax*kmax+j*kmax+(k-1)] )
				+ ( !(threadIdx.x==0) ? sb_t[csb - 1] : p[(i-1)*jmax*kmax+j*kmax+(k-1)] ))
				+ c0[i*jmax*kmax+j*kmax+k] * sb_t[csb]
				+ c1[i*jmax*kmax+j*kmax+k] * (!(threadIdx.y==0) ? sb_m[csb - blockDim.x] : p[i*jmax*kmax+(j-1)*kmax+k])
				+ c2[i*jmax*kmax+j*kmax+k] * (!(threadIdx.x==0) ? sb_m[csb - 1] : p[i*jmax*kmax+j*kmax+(k-1)])
				+ wrk1[i*jmax*kmax+j*kmax+k];

			ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

			temp = temp + ss * ss;

			wrk2[i*jmax*kmax+j*kmax+k] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
    	}
	  	for(i=1 ; i<imax-1 ; i++){
			p[i*jmax*kmax+j*kmax+k] = wrk2[i*jmax*kmax+j*kmax+k];
    	}
  	} /* end n loop */
  	//printf("%d: temp = %d\n", tid, temp);
  	//printf("shared: %f", sb[csb]);
	gosa[tid] = temp;
}

int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[THREAD_NUM];

	/************************************/
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	/************************************/
	mimax = MIMAX;
	mjmax = MJMAX;
	mkmax = MKMAX;
	imax = MIMAX-1;
	jmax = MJMAX-1;
	kmax = MKMAX-1;
	//int N_IJK = MIMAX*MJMAX*MKMAX;
	int N_IJK = mimax*mjmax*mkmax;
	/************************************/
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	/************************************/


	omega = 0.8;

	//initial_maxtrix();

	/******allocate mem on CPU***********/
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*N_IJK);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	/************************************/

	/******allocate mem on GPU***********/
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*THREAD_NUM);
	/************************************/

	/*****Initialize*********************/
	//int i,j,k;

	for(i=0 ; i<mimax ; ++i){
		for(j=0 ; j<mjmax ; ++j){
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				p[i*mjmax*mkmax+j*mkmax+k]=0.0;
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=0.0;
			}
		}
	}

	for(i=0 ; i<mimax ; ++i){
		for(j=0 ; j<mjmax ; ++j){
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				p[i*mjmax*mkmax+j*mkmax+k]=(float)(i*i)/(float)(imax*imax);
				wrk1[i*mjmax*mkmax+j*kmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*kmax+k]=1.0;
			}
		}
	}
	/************************************/

	/*****copy array to device mem*******/
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//cudaMemcpy(dev_gosa, gosa, sizeof(float), cudaMemcpyHostToDevice);
	/************************************/

	printf("mimax = %d mjmax = %d mkmax = %d\n", MIMAX, MJMAX, MKMAX);
	printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	cpu0 = second(); /**measuring**/

	dim3 block(BLOCKSIZEX, BLOCKSIZEY, 1);
	dim3 grid(GRIDSIZEX, GRIDSIZEY, 1);

	jacobi<<<grid, block, sizeof(float) * 3 * BLOCKSIZE>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, NN, mimax, mjmax, mkmax, omega, dev_gosa);

	hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*THREAD_NUM, hipMemcpyDeviceToHost);

	/******Free mem on the GPU**********/
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	/************************************/

	/********Final sum of gosa***********/
	for(int gosa_index=0; gosa_index<THREAD_NUM; gosa_index++){
		final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
	}
	/************************************/

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	if(cpu1 != 0.0)
		xmflops2 = nflop/cpu1*1.0e-6*(float)NN;

	score = xmflops2/32.27;

	printf("gpu: %f sec.\n", cpu1);
	printf("Loop executed for %d times\n", NN);
	printf("Gosa: %e \n", final_gosa);
	printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

	return(0);
}
