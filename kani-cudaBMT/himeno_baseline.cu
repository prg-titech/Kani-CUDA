#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>

<<<<<<< HEAD
#define BLOCKSIZEX 128
#define BLOCKSIZEY 4
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX 4
#define GRIDSIZEY 64
=======
#define BLOCKSIZEX 64
#define BLOCKSIZEY 16
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX 8
#define GRIDSIZEY 16
>>>>>>> b134838f3a58c4117af74290d6fb85e20a9def9b
#define GRIDSIZE GRIDSIZEX * GRIDSIZEY
#define THREAD_NUM BLOCKSIZE * GRIDSIZE

#define MIMAX	256
#define MJMAX	GRIDSIZEY * BLOCKSIZEY + 2
#define MKMAX	GRIDSIZEX * BLOCKSIZEX + 2

#define NN 750

/*static float p[MIMAX][MJMAX][MKMAX];
static float a[MIMAX][MJMAX][MKMAX][4];
static float b[MIMAX][MJMAX][MKMAX][3];
static float c[MIMAX][MJMAX][MKMAX][3];
static float bnd[MIMAX][MJMAX][MKMAX];
static float work1[MIMAX][MJMAX][MKMAX];
static float work2[MIMAX][MJMAX][MKMAX];*/

static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;


/*void initial_matrix(){
	int i, j, k;
	for(i=0; i<imax; i++)
		for(j=0; j<jmax; j++)
			for(k=0; k<kmax; k++){
				a[i][j][k][0] = 0.0;
				a[i][j][k][1] = 0.0;
				a[i][j][k][2] = 0.0;
				a[i][j][k][3] = 0.0;
				b[i][j][k][0] = 0.0;
				b[i][j][k][1] = 0.0;
				b[i][j][k][2] = 0.0;
				c[i][j][k][0] = 0.0;
				c[i][j][k][0]=0.0;
				c[i][j][k][1]=0.0;
				c[i][j][k][2]=0.0;
				p[i][j][k]=0.0;
				work1[i][j][k]=0.0;
				bnd[i][j][k]=0.0;
			}
	for(i=0 ; i<imax ; ++i)
		for(j=0 ; j<jmax ; ++j)
			for(k=0 ; k<kmax ; ++k){
				a[i][j][k][0]=1.0;
				a[i][j][k][1]=1.0;
				a[i][j][k][2]=1.0;
				a[i][j][k][3]=1.0/6.0;
				b[i][j][k][0]=0.0;
				b[i][j][k][1]=0.0;
				b[i][j][k][2]=0.0;
				c[i][j][k][0]=1.0;
				c[i][j][k][1]=1.0;
				c[i][j][k][2]=1.0;
				p[i][j][k]=(float)(k*k)/(float)((kmax-1)*(kmax-1));
				work1[i][j][k]=0.0;
				bnd[i][j][k]=1.0;
			}
}*/

double second(){
	struct timeval tm;
	double t;

	static int base_sec = 0, base_usec = 0;

	gettimeofday(&tm, NULL);

	if(base_sec == 0 && base_usec == 0){
		base_sec = tm.tv_sec;
		base_usec = tm.tv_usec;
		t = 0.0;
	}
	else{
		t = (double)(tm.tv_sec-base_sec) + ((double)(tm.tv_usec-base_usec))/1.0e6;
	}

	return t;
}


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, n, xy, c, csb;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
  k = threadIdx.x + blockDim.x * blockIdx.x + 1;
  j = threadIdx.y + blockDim.y * blockIdx.y + 1;
  const int tid = (k-1) + (j-1) * (kmax-2);
  xy = kmax * jmax;
  __shared__ float sb[BLOCKSIZE];
  csb = threadIdx.x + threadIdx.y * blockDim.x;
  for(n=0;n<nn;++n){
    c = j * kmax + k;
    temp=0.0;
	  for(i=1 ; i<imax-1 ; ++i){
      //sb[csb] = p[c];
      //printf("shared: %f\n", sb[csb]);
      //syncthreads();
			  s0 = a0[i*jmax*kmax+j*kmax+k] * p[(i+1)*jmax*kmax+j*kmax+k]
				+ a1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j+1)*kmax+k]
					   	+ a2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k+1)]
					   	+ b0[i*jmax*kmax+j*kmax+k] * 
					   	( p[(i+1)*jmax*kmax+(j+1)*kmax+k] 
					   - p[(i+1)*jmax*kmax+(j-1)*kmax+k]
						- p[(i-1)*jmax*kmax+(j+1)*kmax+k] + p[(i-1)*jmax*kmax+(j-1)*kmax+k] )
					   + b1[i*jmax*kmax+j*kmax+k] *(p[i*jmax*kmax+(j+1)*kmax+(k+1)] 
					   	- p[i*jmax*kmax+(j-1)*kmax+(k+1)]
						- p[i*jmax*kmax+(j-1)*kmax+(k-1)]
						+ p[i*jmax*kmax+(j+1)*kmax+(k-1)])
					   + b2[i*jmax*kmax+j*kmax+k] * ( p[(i+1)*jmax*kmax+j*kmax+(k+1)] - p[(i-1)*jmax*kmax+j*kmax+(k+1)]
						       - p[(i+1)*jmax*kmax+j*kmax+(k-1)] + p[(i-1)*jmax*kmax+j*kmax+(k-1)] )
					   + c0[i*jmax*kmax+j*kmax+k] * p[(i-1)*jmax*kmax+j*kmax+k]
					   + c1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j-1)*kmax+k]
					   + c2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k-1)]
					   + wrk1[i*jmax*kmax+j*kmax+k];

			 ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

			 temp = temp + ss*ss;

			 wrk2[i*jmax*kmax+j*kmax+k] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
		   c += xy;
    }
	  for(i=1 ; i<imax-1 ; ++i){
				p[i*jmax*kmax+j*kmax+k] = wrk2[i*jmax*kmax+j*kmax+k];
    }
  } /* end n loop */
  //printf("%d: p[%d] = %d\n", tid,i*jmax*kmax+j*kmax+k,p[i*jmax*kmax+j*kmax+k]);
  //printf("shared: %f", sb[csb]);
	gosa[tid] = temp;
}

int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[THREAD_NUM];

	/************************************/
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	/************************************/
  mimax = MIMAX;
  mjmax = MJMAX;
  mkmax = MKMAX;
	imax = MIMAX-1;
	jmax = MJMAX-1;
	kmax = MKMAX-1;
	//int N_IJK = MIMAX*MJMAX*MKMAX;
	int N_IJK = mimax*mjmax*mkmax;
	/************************************/
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	/************************************/


	omega = 0.8;

	//initial_maxtrix();

	/******allocate mem on CPU***********/
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*N_IJK);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	/************************************/

	/******allocate mem on GPU***********/
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*THREAD_NUM);
	/************************************/

	/*****Initialize*********************/
	//int i,j,k;

	for(i=0 ; i<mimax ; ++i)
		for(j=0 ; j<mjmax ; ++j)
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				p[i*mjmax*mkmax+j*mkmax+k]=0.0;
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=0.0;
	      		}

	for(i=0 ; i<mimax ; ++i)
		for(j=0 ; j<mjmax ; ++j)
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				p[i*mjmax*mkmax+j*mkmax+k]=(float)(i*i)/(float)(imax*imax);
				wrk1[i*mjmax*mkmax+j*kmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*kmax+k]=1.0;
			}
	/************************************/

	/*****copy array to device mem*******/
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//hipMemcpy(dev_gosa, gosa, sizeof(float), hipMemcpyHostToDevice);
	/************************************/

	printf("mimax = %d mjmax = %d mkmax = %d\n", MIMAX, MJMAX, MKMAX);
	printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	cpu0 = second(); /**measuring**/

  dim3 block(BLOCKSIZEX, BLOCKSIZEY, 1);
  dim3 grid(GRIDSIZEX, GRIDSIZEY, 1);

	jacobi<<<grid, block>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, NN, mimax, mjmax, mkmax, omega, dev_gosa);

	hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*THREAD_NUM, hipMemcpyDeviceToHost);

	/******Free mem on the GPU**********/
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	/************************************/

	/********Final sum of gosa***********/
	for(int gosa_index=0; gosa_index<THREAD_NUM; gosa_index++){
		final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
	}
	/************************************/

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	if(cpu1 != 0.0)
		xmflops2 = nflop/cpu1*1.0e-6*(float)NN;

	score = xmflops2/32.27;

	printf("gpu: %f sec.\n", cpu1);
	printf("Loop executed for %d times\n", NN);
	printf("Gosa: %e \n", final_gosa);
	printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

	return(0);

}
