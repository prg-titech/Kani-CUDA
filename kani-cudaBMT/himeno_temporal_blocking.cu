
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>

#define BLOCKSIZEX 64
#define BLOCKSIZEY 8
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX 8
#define GRIDSIZEY 45
#define GRIDSIZE GRIDSIZEX * GRIDSIZEY
#define THREAD_NUM BLOCKSIZE * GRIDSIZE

#define MIMAX	256
#define MJMAX	GRIDSIZEY * (BLOCKSIZEY - 2) + 2
#define MKMAX	GRIDSIZEX * (BLOCKSIZEX - 2) + 2

#define NN 700

/*static float p[MIMAX][MJMAX][MKMAX];
static float a[MIMAX][MJMAX][MKMAX][4];
static float b[MIMAX][MJMAX][MKMAX][3];
static float c[MIMAX][MJMAX][MKMAX][3];
static float bnd[MIMAX][MJMAX][MKMAX];
static float work1[MIMAX][MJMAX][MKMAX];
static float work2[MIMAX][MJMAX][MKMAX];*/

static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;

double second(){
	struct timeval tm;
	double t;

	static int base_sec = 0, base_usec = 0;

	gettimeofday(&tm, NULL);

	if(base_sec == 0 && base_usec == 0){
		base_sec = tm.tv_sec;
		base_usec = tm.tv_usec;
		t = 0.0;
	}
	else{
		t = (double)(tm.tv_sec-base_sec) + ((double)(tm.tv_usec-base_usec))/1.0e6;
	}

	return t;
}


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, j2, k2, n, xy, c, csb, csb2;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
	k = threadIdx.x + (blockDim.x-2) * blockIdx.x;
	j = threadIdx.y + (blockDim.y-2) * blockIdx.y;
	k2 = threadIdx.x + blockDim.x * blockIdx.x;
	j2 = threadIdx.y + blockDim.y * blockIdx.y;
	const int tid = (k-1) + (j-1) * (kmax-2);
	xy = kmax * jmax;
	extern __shared__ float sb1[];
  extern __shared__ float sb2[];
  float *sb1_t, *sb1_m, *sb1_b, *sb2_t, *sb2_m, *sb2_b;
  sb1_t = sb1;
  sb1_m = sb1 + (blockDim.x + 2)*(blockDim.y + 2);
  sb1_b = sb1 + 2*(blockDim.x + 2)*(blockDim.y + 2);
  sb2_t = sb2;
  sb2_m = sb2 + BLOCKSIZE;
  sb2_b = sb2 + 2*BLOCKSIZE;
  csb = threadIdx.x + 1 + (threadIdx.y + 1) * (blockDim.x + 2);
	csb2 = threadIdx.x + threadIdx.y * blockDim.x;
  int nn_2 = (int)nn/2;
	for(n=0;n<nn_2;++n){
    i = 0;
		c = j * kmax + k;
		temp=0.0;
    sb1_t[csb] = p[c];
    sb1_m[csb] = p[c+xy];
    sb1_b[csb] = p[c+2*xy];
    if(threadIdx.x==0){
      sb1_t[csb-1] = p[(k==0) ? c : c - 1];
      sb1_m[csb-1] = p[(k==0) ? c+xy : c+xy - 1];
      sb1_b[csb-1] = p[(k==0) ? c+2*xy : c+2*xy - 1];
    }
    if(threadIdx.x==blockDim.x-1){
      sb1_t[csb+1] = p[(k==kmax-1) ? c : c + 1];
      sb1_m[csb+1] = p[(k==kmax-1) ? c+xy : c+xy + 1];
      sb1_b[csb+1] = p[(k==kmax-1) ? c+2*xy : c+2*xy + 1];
    }
    if(threadIdx.y==0){
      sb1_t[csb-blockDim.x-2] = p[(j==0) ? c : c - kmax];
      sb1_m[csb-blockDim.x-2] = p[(j==0) ? c+xy : c+xy - kmax];
      sb1_b[csb-blockDim.x-2] = p[(j==0) ? c+2*xy : c+2*xy - kmax];
    }
    if(threadIdx.y==blockDim.y-1){
      sb1_t[csb+blockDim.x+2] = p[(j==jmax-1) ? c : c + kmax];
      sb1_m[csb+blockDim.x+2] = p[(j==jmax-1) ? c+xy : c+xy + kmax];
      sb1_b[csb+blockDim.x+2] = p[(j==jmax-1) ? c+2*xy : c+2*xy + kmax];
    }
    if(threadIdx.x==0&&threadIdx.y==0){
      sb1_t[0] = p[(k==0||j==0) ? c : c-kmax-1];
      sb1_t[blockDim.x+1] = p[(k+blockDim.x==kmax||j==0) ? c : c-kmax+blockDim.x];
      sb1_t[(blockDim.y+1)*(blockDim.x+2)] = p[(k==0||j+blockDim.y==jmax) ? c : c+blockDim.y*kmax-1];
      sb1_t[(blockDim.y+2)*(blockDim.x+2)-1] = p[(k+blockDim.x==kmax||j+blockDim.y==jmax) ? c : c+blockDim.y*kmax+blockDim.x];
      sb1_m[0] = p[(k==0||j==0) ? c+xy : c+xy-kmax-1];
      sb1_m[blockDim.x+1] = p[(k+blockDim.x==kmax||j==0) ? c+xy : c+xy-kmax+blockDim.x];
      sb1_m[(blockDim.y+1)*(blockDim.x+2)] = p[(k==0||j+blockDim.y==jmax) ? c+xy : c+xy+blockDim.y*kmax-1];
      sb1_m[(blockDim.y+2)*(blockDim.x+2)-1] = p[(k+blockDim.x==kmax||j+blockDim.y==jmax) ? c+xy : c+xy+blockDim.y*kmax+blockDim.x];
      sb1_b[0] = p[(k==0||j==0) ? c+2*xy : c+2*xy-kmax-1];
      sb1_b[blockDim.x+1] = p[(k+blockDim.x==kmax||j==0) ? c+2*xy : c+2*xy-kmax+blockDim.x];
      sb1_b[(blockDim.y+1)*(blockDim.x+2)] = p[(k==0||j+blockDim.y==jmax) ? c+2*xy : c+2*xy+blockDim.y*kmax-1];
      sb1_b[(blockDim.y+2)*(blockDim.x+2)-1] = p[(k+blockDim.x==kmax||j+blockDim.y==jmax) ? c+2*xy : c+2*xy+blockDim.y*kmax+blockDim.x];
    }
    __syncthreads();
    sb2_t[csb2] = sb1_t[csb];
    i = 1;
    s0 =
				a0[i*jmax*kmax+j*kmax+k] * sb1_b[csb]
				+ a1[i*jmax*kmax+j*kmax+k] * sb1_m[csb + blockDim.x + 2]
				+ a2[i*jmax*kmax+j*kmax+k] * sb1_m[csb + 1]
				+ b0[i*jmax*kmax+j*kmax+k] * (
				  sb1_b[csb + blockDim.x + 2]
        - sb1_b[csb - blockDim.x - 2]
        - sb1_t[csb + blockDim.x + 2]
        + sb1_t[csb - blockDim.x - 2])
        + b1[i*jmax*kmax+j*kmax+k] *(
				  sb1_m[csb + blockDim.x + 3]
        - sb1_m[csb - blockDim.x - 1]
        - sb1_m[csb - blockDim.x - 3]
        + sb1_m[csb + blockDim.x + 1])
        + b2[i*jmax*kmax+j*kmax+k] *(
				  sb1_b[csb + 1]
        - sb1_t[csb + 1]
        - sb1_b[csb - 1]
        + sb1_t[csb - 1])
        + c0[i*jmax*kmax+j*kmax+k] * sb1_t[csb]
				+ c1[i*jmax*kmax+j*kmax+k] * sb1_m[csb - blockDim.x - 2]
				+ c2[i*jmax*kmax+j*kmax+k] * sb1_m[csb - 1]
				+ wrk1[i*jmax*kmax+j*kmax+k];

		ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - sb1_m[csb] ) * bnd[i*jmax*kmax+j*kmax+k];

		temp = temp + ss*ss;

		wrk2[i*jmax*kmax+j*kmax+k] = (k==0||k==kmax-1||j==0||j==jmax-1) ? sb1_m[csb] : (sb1_m[csb] + omega * ss);

    sb2_m[csb2] = wrk2[i*jmax*kmax+j*kmax+k];
		c += 2*xy;
    __syncthreads();

		for(i=2 ; i<imax-1 ; ++i){
      float *sb_tmp = sb1_t;
      sb1_t = sb1_m;
      sb1_m = sb1_b;
      sb1_b = sb_tmp;
      sb1_b[csb] = p[c+xy];
      if(threadIdx.x == 0){ sb1_b[csb-1] = p[(k==0) ? c+xy : c+xy-1];}
      if(threadIdx.x == blockDim.x-1){ sb1_b[csb+1] = p[(k==kmax-1) ? c+xy : c+xy+1];}
      if(threadIdx.y == 0){ sb1_b[csb-blockDim.x-2] = p[(j==0) ? c+xy : c+xy-kmax];}
      if(threadIdx.y == blockDim.y-1){ sb1_b[csb+blockDim.x+2] = p[(j==jmax-1) ? c+xy : c+xy+kmax];}
			if(threadIdx.x == 0 && threadIdx.y == 0){
        sb1_b[0] = p[(k==0||j==0) ? c+xy : c+xy-kmax-1];
        sb1_b[blockDim.x+2] = p[(k+blockDim.x==kmax||j==0) ? c+xy : c+xy-kmax+blockDim.x];
        sb1_b[(blockDim.y+1)*(blockDim.x+2)] = p[(k==0||j+blockDim.y==jmax) ? c+xy : c+xy+blockDim.y*kmax-1];
        sb1_b[(blockDim.y+2)*(blockDim.x+2)] = p[(k+blockDim.x==kmax||j+blockDim.y==jmax) ? c+xy : c+xy+blockDim.y*kmax+blockDim.x];
      }
      __syncthreads();

      s0 =
				a0[i*jmax*kmax+j*kmax+k] * sb1_b[csb]
				+ a1[i*jmax*kmax+j*kmax+k] * sb1_m[csb + blockDim.x + 2]
				+ a2[i*jmax*kmax+j*kmax+k] * sb1_m[csb + 1]
				+ b0[i*jmax*kmax+j*kmax+k] * (
				  sb1_b[csb + blockDim.x + 2]
        - sb1_b[csb - blockDim.x - 2]
        - sb1_t[csb + blockDim.x + 2]
        + sb1_t[csb - blockDim.x - 2])
        + b1[i*jmax*kmax+j*kmax+k] *(
				  sb1_m[csb + blockDim.x + 3]
        - sb1_m[csb - blockDim.x - 1]
        - sb1_m[csb - blockDim.x - 3]
        + sb1_m[csb + blockDim.x + 1])
        + b2[i*jmax*kmax+j*kmax+k] *(
				  sb1_b[csb + 1]
        - sb1_t[csb + 1]
        - sb1_b[csb - 1]
        + sb1_t[csb - 1])
        + c0[i*jmax*kmax+j*kmax+k] * sb1_t[csb]
				+ c1[i*jmax*kmax+j*kmax+k] * sb1_m[csb - blockDim.x - 2]
				+ c2[i*jmax*kmax+j*kmax+k] * sb1_m[csb - 1]
				+ wrk1[i*jmax*kmax+j*kmax+k];

		  ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - sb1_b[csb] ) * bnd[i*jmax*kmax+j*kmax+k];

		  temp = temp + ss*ss;

		  wrk2[i*jmax*kmax+j*kmax+k] = (k==0||k==kmax-1||j==0||j==jmax-1) ? sb1_m[csb] : (sb1_b[csb] + omega * ss);

      sb2_b[csb2] = wrk2[i*jmax*kmax+j*kmax+k];

	  	__syncthreads();

      if(0<threadIdx.x && threadIdx.x<blockDim.x-1 && 0<threadIdx.y && threadIdx.y<blockDim.y-1){
		    s0 =
				a0[i*jmax*kmax+j*kmax+k] * sb2_b[csb2]
				+ a1[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 + blockDim.x]
				+ a2[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 + 1]
				+ b0[i*jmax*kmax+j*kmax+k] * (
				  sb2_b[csb2 + blockDim.x]
        - sb2_b[csb2 - blockDim.x]
        - sb2_t[csb2 + blockDim.x]
        + sb2_t[csb2 - blockDim.x])
        + b1[i*jmax*kmax+j*kmax+k] *(
				  sb2_m[csb2 + blockDim.x]
        - sb2_m[csb2 - blockDim.x]
        - sb2_m[csb2 - blockDim.x]
        + sb2_m[csb2 + blockDim.x])
        + b2[i*jmax*kmax+j*kmax+k] *(
				  sb2_b[csb2 + 1]
        - sb2_t[csb2 + 1]
        - sb2_b[csb2 - 1]
        + sb2_t[csb2 - 1])
        + c0[i*jmax*kmax+j*kmax+k] * sb2_t[csb2]
				+ c1[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 - blockDim.x]
				+ c2[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 - 1]
				+ wrk1[i*jmax*kmax+j*kmax+k];


		  ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - sb2_m[csb2] ) * bnd[i*jmax*kmax+j*kmax+k];

	  	temp = temp + ss*ss;

		  p[i*jmax*kmax+j*kmax+k] = sb2_m[csb2] + omega * ss;
      }
		c += xy;
    __syncthreads();

    sb_tmp = sb2_t;
    sb2_t = sb2_m;
    sb2_m = sb2_b;
    sb2_b = sb_tmp;
		}


    sb2_b[csb2] = sb1_b[csb];
    if(0<threadIdx.x && threadIdx.x<blockDim.x-1 && 0<threadIdx.y && threadIdx.y<blockDim.y-1){
		  s0 =
				a0[i*jmax*kmax+j*kmax+k] * sb2_b[csb2]
				+ a1[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 + blockDim.x]
				+ a2[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 + 1]
				+ b0[i*jmax*kmax+j*kmax+k] * (
				  sb2_b[csb2 + blockDim.x]
        - sb2_b[csb2 - blockDim.x]
        - sb2_t[csb2 + blockDim.x]
        + sb2_t[csb2 - blockDim.x])
        + b1[i*jmax*kmax+j*kmax+k] *(
				  sb2_m[csb2 + blockDim.x]
        - sb2_m[csb2 - blockDim.x]
        - sb2_m[csb2 - blockDim.x]
        + sb2_m[csb2 + blockDim.x])
        + b2[i*jmax*kmax+j*kmax+k] *(
				  sb2_b[csb2 + 1]
        - sb2_t[csb2 + 1]
        - sb2_b[csb2 - 1]
        + sb2_t[csb2 - 1])
        + c0[i*jmax*kmax+j*kmax+k] * sb2_t[csb2]
				+ c1[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 - blockDim.x]
				+ c2[i*jmax*kmax+j*kmax+k] * sb2_m[csb2 - 1]
				+ wrk1[i*jmax*kmax+j*kmax+k];


		  ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - sb2_m[csb2] ) * bnd[i*jmax*kmax+j*kmax+k];

	  	temp = temp + ss*ss;

		  p[i*jmax*kmax+j*kmax+k] = sb2_m[csb2] + omega * ss;
    }
    __syncthreads();
  } /* end n loop */
	__syncthreads();
	gosa[tid] = temp;
}

int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[THREAD_NUM];

	/************************************/
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	/************************************/
	mimax = MIMAX;
	mjmax = MJMAX;
	mkmax = MKMAX;
	imax = MIMAX-1;
	jmax = MJMAX-1;
	kmax = MKMAX-1;
	//int N_IJK = MIMAX*MJMAX*MKMAX;
	int N_IJK = mimax*mjmax*mkmax;
	int WORKSIZE = THREAD_NUM*mimax;
	/************************************/
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	/************************************/


	omega = 0.8;

	//initial_maxtrix();

	/******allocate mem on CPU***********/
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*WORKSIZE);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	/************************************/

	/******allocate mem on GPU***********/
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, WORKSIZE*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*THREAD_NUM);
	/************************************/

	/*****Initialize*********************/
	//int i,j,k;
	/*
	for(i=0 ; i<mimax ; ++i)
		for(j=0 ; j<mjmax ; ++j)
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				p[i*mjmax*mkmax+j*mkmax+k]=0.0;
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=0.0;
	      		}
	*/
	for(i=0 ; i<mimax ; ++i){
		for(j=0 ; j<mjmax ; ++j){
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				p[i*mjmax*mkmax+j*mkmax+k]=(float)(i*i)/(float)(imax*imax);
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
        wrk2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=1.0;
			}
		}
	}
	/************************************/

	/*****copy array to device mem*******/
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//cudaMemcpy(dev_gosa, gosa, sizeof(float), cudaMemcpyHostToDevice);
	/************************************/

	printf("mimax = %d mjmax = %d mkmax = %d\n", MIMAX, MJMAX, MKMAX);
	printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	cpu0 = second(); /**measuring**/

	dim3 block(BLOCKSIZEX, BLOCKSIZEY, 1);
	dim3 grid(GRIDSIZEX, GRIDSIZEY, 1);

	jacobi<<<grid, block, sizeof(float)*3*(BLOCKSIZE+(BLOCKSIZEX+2)*(BLOCKSIZEY+2))>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, NN, mimax, mjmax, mkmax, omega, dev_gosa);

  //for(int i = 0; i<N_IJK; i++){
  //  printf("%f ", p[i]);
  //}
	hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*THREAD_NUM, hipMemcpyDeviceToHost);

	/******Free mem on the GPU**********/
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	/************************************/

	/********Final sum of gosa***********/
	for(int gosa_index=0; gosa_index<THREAD_NUM; gosa_index++){
		//printf("%f\n", gosa[gosa_index]);
		final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
	}
	/************************************/

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	if(cpu1 != 0.0){
		xmflops2 = nflop/cpu1*1.0e-6*(float)NN;
	}

	score = xmflops2/32.27;

	printf("gpu: %f sec.\n", cpu1);
	printf("Loop executed for %d times\n", NN);
	printf("Gosa: %e \n", final_gosa);
	printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

	return(0);
}
