
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>

#define BLOCKSIZEX 6
#define BLOCKSIZEY 4
#define BLOCKSIZE BLOCKSIZEX * BLOCKSIZEY
#define GRIDSIZEX 3
#define GRIDSIZEY 6
#define GRIDSIZE GRIDSIZEX * GRIDSIZEY
#define THREAD_NUM BLOCKSIZE * GRIDSIZE

#define MIMAX	4
#define MJMAX	GRIDSIZEY * (BLOCKSIZEY - 2) + 2
#define MKMAX	GRIDSIZEX * (BLOCKSIZEX - 2) + 2

#define NN 3

/*static float p[MIMAX][MJMAX][MKMAX];
static float a[MIMAX][MJMAX][MKMAX][4];
static float b[MIMAX][MJMAX][MKMAX][3];
static float c[MIMAX][MJMAX][MKMAX][3];
static float bnd[MIMAX][MJMAX][MKMAX];
static float work1[MIMAX][MJMAX][MKMAX];
static float work2[MIMAX][MJMAX][MKMAX];*/

static int imax, jmax, kmax, mimax, mjmax, mkmax;
static float omega;

double second(){
	struct timeval tm;
	double t;

	static int base_sec = 0, base_usec = 0;

	gettimeofday(&tm, NULL);

	if(base_sec == 0 && base_usec == 0){
		base_sec = tm.tv_sec;
		base_usec = tm.tv_usec;
		t = 0.0;
	}
	else{
		t = (double)(tm.tv_sec-base_sec) + ((double)(tm.tv_usec-base_usec))/1.0e6;
	}

	return t;
}


__global__ void jacobi(float *a0, float *a1, float *a2, float *a3, float *b0, float *b1, float *b2, float *c0, float *c1, float *c2, float *p, float *wrk1, float *wrk2, float *bnd, int nn, int imax, int jmax, int kmax, float omega, float *gosa){
	int i, j, k, j2, k2, n, xy, c, csb;
	float s0, ss, temp;
	//const int size = (imax-1)/(imax-1);
	k = threadIdx.x + (blockDim.x-2) * blockIdx.x + 1;
	j = threadIdx.y + (blockDim.y-2) * blockIdx.y + 1;
	k2 = threadIdx.x + blockDim.x * blockIdx.x;
	j2 = threadIdx.y + blockDim.y * blockIdx.y;
	const int tid = (k-1) + (j-1) * (kmax-2);
	xy = kmax * jmax;
	__shared__ float sb[BLOCKSIZE];
	csb = threadIdx.x + threadIdx.y * blockDim.x;
	for(n=0;n<nn;++n){
		c = j * kmax + k;
		temp=0.0;
		if(0 < threadIdx.x && k < kmax-1 && 0 < j && j < jmax-1){
			for(i=1 ; i<imax-1 ; ++i){
				s0 = a0[i*jmax*kmax+j*kmax+k] * p[(i+1)*jmax*kmax+j*kmax+k]
				+ a1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j+1)*kmax+k]
				+ a2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k+1)]
				+ b0[i*jmax*kmax+j*kmax+k]
					*(p[(i+1)*jmax*kmax+(j+1)*kmax+k]
					- p[(i+1)*jmax*kmax+(j-1)*kmax+k]
					- p[(i-1)*jmax*kmax+(j+1)*kmax+k]
					+ p[(i-1)*jmax*kmax+(j-1)*kmax+k] )
				+ b1[i*jmax*kmax+j*kmax+k]
					*(p[i*jmax*kmax+(j+1)*kmax+(k+1)]
					- p[i*jmax*kmax+(j-1)*kmax+(k+1)]
					- p[i*jmax*kmax+(j-1)*kmax+(k-1)]
					+ p[i*jmax*kmax+(j+1)*kmax+(k-1)])
				+ b2[i*jmax*kmax+j*kmax+k]
					*(p[(i+1)*jmax*kmax+j*kmax+(k+1)]
					- p[(i-1)*jmax*kmax+j*kmax+(k+1)]
					- p[(i+1)*jmax*kmax+j*kmax+(k-1)]
					+ p[(i-1)*jmax*kmax+j*kmax+(k-1)] )
				+ c0[i*jmax*kmax+j*kmax+k] * p[(i-1)*jmax*kmax+j*kmax+k]
				+ c1[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+(j-1)*kmax+k]
				+ c2[i*jmax*kmax+j*kmax+k] * p[i*jmax*kmax+j*kmax+(k-1)]
				+ wrk1[i*jmax*kmax+j*kmax+k];

				ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - p[i*jmax*kmax+j*kmax+k] ) * bnd[i*jmax*kmax+j*kmax+k];

				temp = temp + ss*ss;

				wrk2[i*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+k2] = p[i*jmax*kmax+j*kmax+k] + omega * ss;
				c += xy;
			}
		}
		__syncthreads();
		if(0 < threadIdx.x && threadIdx.x < blockDim.x-1 && 0 < threadIdx.y && threadIdx.y < blockDim.y-1){
			for(i=1; i<imax-1; i++){
				s0 = a0[i*jmax*kmax+j*kmax+k] * wrk2[(i+1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+k2]
				+ a1[i*jmax*kmax+j*kmax+k] * wrk2[i*THREAD_NUM+(j2+1)*BLOCKSIZEX*GRIDSIZEX+k2]
				+ a2[i*jmax*kmax+j*kmax+k] * wrk2[i*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2+1)]
				+ b0[i*jmax*kmax+j*kmax+k]
					*(wrk2[(i+1)*THREAD_NUM+(j2+1)*BLOCKSIZEX*GRIDSIZEX+k2]
					- wrk2[(i+1)*THREAD_NUM+(j2-1)*BLOCKSIZEX*GRIDSIZEX+k2]
					- wrk2[(i-1)*THREAD_NUM+(j2+1)*BLOCKSIZEX*GRIDSIZEX+k2]
					+ wrk2[(i-1)*THREAD_NUM+(j2-1)*BLOCKSIZEX*GRIDSIZEX+k2] )
				+ b1[i*jmax*kmax+j*kmax+k]
					*(wrk2[i*THREAD_NUM+(j2+1)*BLOCKSIZEX*GRIDSIZEX+(k2+1)]
					- wrk2[i*THREAD_NUM+(j2-1)*BLOCKSIZEX*GRIDSIZEX+(k2+1)]
					- wrk2[i*THREAD_NUM+(j2-1)*BLOCKSIZEX*GRIDSIZEX+(k2-1)]
					+ wrk2[i*THREAD_NUM+(j2+1)*BLOCKSIZEX*GRIDSIZEX+(k2-1)])
				+ b2[i*jmax*kmax+j*kmax+k]
					*(wrk2[(i+1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2+1)]
					- wrk2[(i-1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2+1)]
					- wrk2[(i+1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2-1)]
					+ wrk2[(i-1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2-1)] )
				+ c0[i*jmax*kmax+j*kmax+k] * wrk2[(i-1)*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+k2]
				+ c1[i*jmax*kmax+j*kmax+k] * wrk2[i*THREAD_NUM+(j2-1)*BLOCKSIZEX*GRIDSIZEX+k2]
				+ c2[i*jmax*kmax+j*kmax+k] * wrk2[i*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+(k2-1)]
				+ wrk1[i*jmax*kmax+j*kmax+k];

				ss = ( s0 * a3[i*jmax*kmax+j*kmax+k] - wrk2[i*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+k2] ) * bnd[i*jmax*kmax+j*kmax+k];

				temp = temp + ss*ss;

				p[i*jmax*kmax+j*kmax+k] = wrk2[i*THREAD_NUM+j2*BLOCKSIZEX*GRIDSIZEX+k2] + omega * ss;
				c += xy;
			}
		}
	} /* end n loop */
	__syncthreads();
	gosa[tid] = temp;
}

int main(){
	int i, j, k;
	float final_gosa;
	double cpu0, cpu1, nflop, xmflops2, score;

	float gosa[THREAD_NUM];

	/************************************/
	float *p;
	float *a0, *a1, *a2, *a3;
	float *b0, *b1, *b2;
	float *c0, *c1, *c2;

	float *bnd;
	float *wrk1, *wrk2;
	/************************************/
	mimax = MIMAX;
	mjmax = MJMAX;
	mkmax = MKMAX;
	imax = MIMAX-1;
	jmax = MJMAX-1;
	kmax = MKMAX-1;
	//int N_IJK = MIMAX*MJMAX*MKMAX;
	int N_IJK = mimax*mjmax*mkmax;
	int WORKSIZE = THREAD_NUM*mimax;
	/************************************/
	float *dev_p;
	float *dev_a0, *dev_a1, *dev_a2, *dev_a3;
	float *dev_b0, *dev_b1, *dev_b2;
	float *dev_c0, *dev_c1, *dev_c2;

	float *dev_bnd;
	float *dev_wrk1, *dev_wrk2;

	float *dev_gosa;
	/************************************/


	omega = 0.8;

	//initial_maxtrix();

	/******allocate mem on CPU***********/
	a0 = (float*)malloc(sizeof(float)*N_IJK);
	a1 = (float*)malloc(sizeof(float)*N_IJK);
	a2 = (float*)malloc(sizeof(float)*N_IJK);
	a3 = (float*)malloc(sizeof(float)*N_IJK);

	b0 = (float*)malloc(sizeof(float)*N_IJK);
	b1 = (float*)malloc(sizeof(float)*N_IJK);
	b2 = (float*)malloc(sizeof(float)*N_IJK);

	c0 = (float*)malloc(sizeof(float)*N_IJK);
	c1 = (float*)malloc(sizeof(float)*N_IJK);
	c2 = (float*)malloc(sizeof(float)*N_IJK);

	p = (float*)malloc(sizeof(float)*N_IJK);

	wrk1 = (float*)malloc(sizeof(float)*N_IJK);
	wrk2 = (float*)malloc(sizeof(float)*WORKSIZE);
	bnd = (float*)malloc(sizeof(float)*N_IJK);

	//gosa = (float*)malloc(sizeof(float));
	/************************************/

	/******allocate mem on GPU***********/
	hipMalloc((void**)&dev_a0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a2, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_a3, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_b0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_b2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_c0, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_c2, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_p, N_IJK*sizeof(float));

	hipMalloc((void**)&dev_bnd, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk1, N_IJK*sizeof(float));
	hipMalloc((void**)&dev_wrk2, WORKSIZE*sizeof(float));

	hipMalloc((void**)&dev_gosa, sizeof(float)*THREAD_NUM);
	/************************************/

	/*****Initialize*********************/
	//int i,j,k;
	/*
	for(i=0 ; i<mimax ; ++i)
		for(j=0 ; j<mjmax ; ++j)
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				p[i*mjmax*mkmax+j*mkmax+k]=0.0;
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=0.0;
	      		}
	*/
	for(i=0 ; i<mimax ; ++i){
		for(j=0 ; j<mjmax ; ++j){
			for(k=0 ; k<mkmax ; ++k){
				a0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				a3[i*mjmax*mkmax+j*mkmax+k]=1.0/6.0;
				b0[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				b2[i*mjmax*mkmax+j*mkmax+k]=0.0;
				c0[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c1[i*mjmax*mkmax+j*mkmax+k]=1.0;
				c2[i*mjmax*mkmax+j*mkmax+k]=1.0;
				p[i*mjmax*mkmax+j*mkmax+k]=(float)(i*i)/(float)(imax*imax);
				wrk1[i*mjmax*mkmax+j*mkmax+k]=0.0;
				bnd[i*mjmax*mkmax+j*mkmax+k]=1.0;
			}
		}
	}
	/************************************/

	/*****copy array to device mem*******/
	hipMemcpy(dev_a0, a0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a1, a1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a2, a2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a3, a3, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b0, b0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b1, b1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b2, b2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c0, c0, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c1, c1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c2, c2, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk1, wrk1, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_wrk2, wrk2, WORKSIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bnd, bnd, N_IJK*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_p, p, N_IJK*sizeof(float), hipMemcpyHostToDevice);

	//cudaMemcpy(dev_gosa, gosa, sizeof(float), cudaMemcpyHostToDevice);
	/************************************/

	printf("mimax = %d mjmax = %d mkmax = %d\n", MIMAX, MJMAX, MKMAX);
	printf("imax = %d jmax = %d kmax = %d\n", imax, jmax, kmax);

	cpu0 = second(); /**measuring**/

	dim3 block(BLOCKSIZEX, BLOCKSIZEY, 1);
	dim3 grid(GRIDSIZEX, GRIDSIZEY, 1);

	jacobi<<<grid, block>>>(dev_a0, dev_a1, dev_a2, dev_a3, dev_b0, dev_b1, dev_b2, dev_c0, dev_c1, dev_c2, dev_p, dev_wrk1, dev_wrk2, dev_bnd, NN, mimax, mjmax, mkmax, omega, dev_gosa);

	hipDeviceSynchronize();

	cpu1 = second();

	hipMemcpy(&gosa, dev_gosa, sizeof(float)*THREAD_NUM, hipMemcpyDeviceToHost);

	/******Free mem on the GPU**********/
	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_a2);
	hipFree(dev_a3);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_b2);
	hipFree(dev_c0);
	hipFree(dev_c1);
	hipFree(dev_c2);
	hipFree(dev_p);
	hipFree(dev_wrk1);
	hipFree(dev_wrk2);
	hipFree(dev_bnd);

	hipFree(dev_gosa);
	/************************************/

	/********Final sum of gosa***********/
	for(int gosa_index=0; gosa_index<THREAD_NUM; gosa_index++){
		//printf("%f\n", gosa[gosa_index]);
		final_gosa += gosa[gosa_index];
		//printf("Gosa%d: %e \n", gosa_index, gosa[gosa_index]);
	}
	/************************************/

	nflop = (kmax-2)*(jmax-2)*(imax-2)*34;

	if(cpu1 != 0.0){
		xmflops2 = nflop/cpu1*1.0e-6*(float)NN;
	}

	score = xmflops2/32.27;

	printf("gpu: %f sec.\n", cpu1);
	printf("Loop executed for %d times\n", NN);
	printf("Gosa: %e \n", final_gosa);
	//printf("MFLOPS measured: %f\n", xmflops2);
	//printf("Score: %f\n", score);

	return(0);
}
