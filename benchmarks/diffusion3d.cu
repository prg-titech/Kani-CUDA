#include "hip/hip_runtime.h"
#include<stdio.h>
#include "stopwatch.h"

#define BLOCK_X 3
#define BLOCK_Y 4
#define GRID_X 3
#define GRID_Y 3
//#define M_PI (3.1415926535897932384626)

__global__ void diffusion_kernel(float* in,
                                 float* out,
                                 int nx, int ny, int nz,
                                 float ce, float cw, float cn, float cs,
                                 float ct, float cb, float cc) {
  profile("threadIdx.x threadIdx.y blockDim.x blockDim.y csb c i j");
  int i = blockDim.x * blockIdx.x + threadIdx.x;  
  int j = blockDim.y * blockIdx.y + threadIdx.y;
  int c = i + j * nx;
  int xy = nx * ny;
  for (int k = 0; k < nz; ++k) {
    sb[csb] = in[c];
    int w = (i == 0)        ? c : c - 1;
    int e = (i == nx-1)     ? c : c + 1;
    int n = (j == 0)        ? c : c - nx;
    int s = (j == ny-1)     ? c : c + nx;
    int b = (k == 0)        ? c : c - xy;
    int t = (k == nz-1)     ? c : c + xy;
    out[c] = 
        cc * in[c] 
      + cw * in[w] 
      + ce * in[e] 
      + cs * in[s]
      + cn * in[n] 
      + cb * in[b] 
      + ct * in[t];   
    c += xy;
  }
}

void initialize(float *buff, const int nx, const int ny, const int nz,
                const float kx, const float ky, const float kz,
                const float dx, const float dy, const float dz,
                const float kappa, const float time) {
  float ax = exp(-kappa*time*(kx*kx));
  float ay = exp(-kappa*time*(ky*ky));
  float az = exp(-kappa*time*(kz*kz));
  int jz;  
  for (jz = 0; jz < nz; jz++) {
    int jy;
    for (jy = 0; jy < ny; jy++) {
      int jx;
      for (jx = 0; jx < nx; jx++) {
        int j = jz*nx*ny + jy*nx + jx;
        float x = dx*((float)(jx + 0.5));
        float y = dy*((float)(jy + 0.5));
        float z = dz*((float)(jz + 0.5));
        float f0 = (float)0.125
          *(1.0 - ax*cos(kx*x))
          *(1.0 - ay*cos(ky*y))
          *(1.0 - az*cos(kz*z));
        buff[j] = f0;
      }
    }
  }
}


int main(){
  int count = 5;
  int nx, ny, nz;
  nx = BLOCK_X*GRID_X;
  ny = BLOCK_Y*GRID_Y;
  nz = 4;

  float l, kappa;
  float kx, ky, kz;
  float dx, dy, dz, dt;
  float ce, cw, cn, cs, ct, cb, cc;
  l = 1.0;
  kappa = 0.1;
  dx = l / nx;
  dy = l / ny;
  dz = l / nz;
  kx = 2.0 * M_PI;
  ky = 2.0 * M_PI;
  kz = 2.0 * M_PI;
  dt = 0.1 * dx * dy / kappa;
  ce = kappa * dt /(dx*dx);
  cw = kappa * dt /(dx*dx);
  cn = kappa * dt /(dy*dy);
  cs = kappa * dt /(dy*dy);
  ct = kappa * dt /(dz*dz);
  cb = kappa * dt /(dz*dz);
  cc = 1.0 - (ce+cw+cn+cs+ct+cb);

  float *in, *dev_in, *dev_out;
  int s = sizeof(float) * nx * ny * nz;
  in = (float *)malloc(s);
  initialize(in, nx, ny, nz,
             kx, ky, kz, dx, dy, dz,
             kappa, 0.0);
  hipMalloc((void**)&dev_in, s);
  hipMalloc((void**)&dev_out, s);
  hipMemcpy(dev_in, in, s, hipMemcpyHostToDevice);


  dim3 block(BLOCK_X, BLOCK_Y, 1);
  dim3 grid(GRID_X, GRID_Y, 1);

  for(int k=0; k<nz; k++){
    for(int j=0; j<ny; j++){
      for(int i=0; i<nx; i++){
        printf("%f ", in[i+j*nx+k*nx*ny]);
      }
      printf("\n");
    }
    printf("\n");
  }

  Stopwatch st;
  StopwatchStart(&st);

  for(int i = 0; i < count; i++){
    diffusion_kernel<<<grid, block>>>(
      dev_in, dev_out, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
    float *t;
    t = dev_in;
    dev_in = dev_out;
    dev_out = t;
  }
  hipMemcpy(in, dev_in, s, hipMemcpyDeviceToHost);

  float elapsed_time = StopwatchStop(&st);

  printf("pass\n", elapsed_time);
  printf("kernel time: %f\n", elapsed_time);
  
  for(int k=0; k<nz; k++){
    for(int j=0; j<ny; j++){
      for(int i=0; i<nx; i++){
        printf("%f ", in[i+j*nx+k*nx*ny]);
      }
      printf("\n");
    }
    printf("\n");
  }
  
  
  return(0);
}